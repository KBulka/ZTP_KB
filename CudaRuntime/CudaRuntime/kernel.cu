#include "hip/hip_runtime.h"
﻿#include <windows.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#pragma pack(push, 1)
typedef struct {
    unsigned short type;
    unsigned int size;
    unsigned short reserved1;
    unsigned short reserved2;
    unsigned int offset;
} BMPHeader;

typedef struct {
    unsigned int size;
    int width;
    int height;
    unsigned short planes;
    unsigned short bitCount;
    unsigned int compression;
    unsigned int sizeImage;
    int xPelsPerMeter;
    int yPelsPerMeter;
    unsigned int clrUsed;
    unsigned int clrImportant;
} BMPInfoHeader;
#pragma pack(pop)

// CUDA kernel: invert pixel colors
__global__ void invertKernel(unsigned char* data, int dataSize) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < dataSize) {
        data[i] = 255 - data[i];
    }
}

extern "C" __declspec(dllexport)
void InvertImage(const char* inputPath, const char* outputPath)
{
    FILE* fp = fopen(inputPath, "rb");
    if (!fp) {
        printf("Nie można otworzyć pliku wejściowego: %s\n", inputPath);
        return;
    }

    BMPHeader header;
    BMPInfoHeader info;
    fread(&header, sizeof(BMPHeader), 1, fp);
    fread(&info, sizeof(BMPInfoHeader), 1, fp);

    if (info.bitCount != 24) {
        printf("Obsługiwane są tylko pliki BMP 24-bitowe!\n");
        fclose(fp);
        return;
    }

    int imageSize = info.width * info.height * 3;
    unsigned char* img = (unsigned char*)malloc(imageSize);
    fseek(fp, header.offset, SEEK_SET);
    fread(img, 1, imageSize, fp);
    fclose(fp);

    // CUDA memory and execution
    unsigned char* d_img;
    hipMalloc(&d_img, imageSize);
    hipMemcpy(d_img, img, imageSize, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (imageSize + threads - 1) / threads;
    invertKernel << <blocks, threads >> > (d_img, imageSize);
    hipDeviceSynchronize();

    hipMemcpy(img, d_img, imageSize, hipMemcpyDeviceToHost);
    hipFree(d_img);

    // Zapis wyniku
    FILE* out = fopen(outputPath, "wb");
    if (!out) {
        printf("Nie można otworzyć pliku wyjściowego: %s\n", outputPath);
        free(img);
        return;
    }

    fwrite(&header, sizeof(BMPHeader), 1, out);
    fwrite(&info, sizeof(BMPInfoHeader), 1, out);
    fseek(out, header.offset, SEEK_SET);
    fwrite(img, 1, imageSize, out);
    fclose(out);

    free(img);
    printf("Zapisano: %s\n", outputPath);
}
